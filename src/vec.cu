#include "hip/hip_runtime.h"
//
//  vec.cpp
//  CUDA Physics
//
//  Created by Jacob Austin on 5/13/18.
//  Copyright © 2018 Jacob Austin. All rights reserved.
//

#include "vec.h"
#include <hip/hip_complex.h>

#if __CUDA_ARCH__ < 600
__device__ double atomicDoubleAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
						__double_as_longlong(val +
											 __longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}

__device__ double atomicDoubleExch(double* address, double val)
{
	unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
						__double_as_longlong(val));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif

CUDA_DEVICE void Vec::atomicVecAdd(const Vec & v) {
atomicDoubleAdd(&data[0], (double) v.data[0]);
atomicDoubleAdd(&data[1], (double) v.data[1]);
atomicDoubleAdd(&data[2], (double) v.data[2]);
}

CUDA_DEVICE void Vec::atomicVecExch(const Vec &v) {
	atomicDoubleExch(&data[0], (double) v.data[0]);
	atomicDoubleExch(&data[1], (double) v.data[1]);
	atomicDoubleExch(&data[2], (double) v.data[2]);
}

CUDA_CALLABLE_MEMBER Vec::Vec() {
		data[0] = 0;
		data[1] = 0;
		data[2] = 0;
	} // default

CUDA_CALLABLE_MEMBER Vec::Vec(const Vec & v) {
	data[0] = v.data[0];
	data[1] = v.data[1];
	data[2] = v.data[2];
} // copy constructor

CUDA_CALLABLE_MEMBER Vec::Vec(double x, double y, double z) {
	data[0] = x;
	data[1] = y;
	data[2] = z;
} // initialization from x, y, and z values

CUDA_CALLABLE_MEMBER Vec & Vec::operator=(const Vec & v) {
	if (this == &v) {
		return *this;
	}

	data[0] = v.data[0];
	data[1] = v.data[1];
	data[2] = v.data[2];

	return *this;
}

CUDA_CALLABLE_MEMBER Vec & Vec::operator+=(const Vec & v) {
	data[0] += v.data[0];
	data[1] += v.data[1];
	data[2] += v.data[2];
	return *this;
}

CUDA_CALLABLE_MEMBER Vec & Vec::operator-=(const Vec & v) {
	data[0] -= v.data[0];
	data[1] -= v.data[1];
	data[2] -= v.data[2];
	return *this;
}

CUDA_DEVICE void Vec::atomicVecAdd(const Vec & v);
CUDA_DEVICE void Vec::atomicVecExch(const Vec &v);

CUDA_CALLABLE_MEMBER Vec Vec::operator-() const{
	return Vec(-data[0], -data[1], -data[2]);
}

CUDA_CALLABLE_MEMBER double & Vec::operator [] (int n) {
	if (n < 0 || n >= 3) {
		printf("%s\n", "Out of bounds!");
		return data[0];
	} else {
		return data[n];
	}
}

CUDA_CALLABLE_MEMBER const double & Vec::operator [] (int n) const {
	if (n < 0 || n >= 3) {
		printf("%s\n", "Out of bounds!");
		return data[0];
	} else {
		return data[n];
	}
}

CUDA_CALLABLE_MEMBER friend Vec Vec::operator+(const Vec & v1, const Vec & v2) {
	return Vec(v1.data[0] + v2.data[0], v1.data[1] + v2.data[1], v1.data[2] + v2.data[2]);
}

CUDA_CALLABLE_MEMBER friend Vec Vec::operator-(const Vec & v1, const Vec & v2) {
	return Vec(v1.data[0] - v2.data[0], v1.data[1] - v2.data[1], v1.data[2] - v2.data[2]);
}

CUDA_CALLABLE_MEMBER friend Vec Vec::operator*(const double x, const Vec & v) {
	return Vec(v.data[0] * x, v.data[1] * x, v.data[2] * x);
}

CUDA_CALLABLE_MEMBER friend Vec Vec::operator*(const Vec & v, const double x) {
	return x * v;
} // double times Vec

CUDA_CALLABLE_MEMBER friend bool Vec::operator==(const Vec & v1, const Vec & v2) {
	return (v1[0] == v2[0] && v1[1] == v2[1] && v1[2] == v2[2]);
}

CUDA_CALLABLE_MEMBER friend bool Vec::operator<(const Vec &v1, const Vec &v2) {
	return ((v1[0] < v2[0]) || (v1[0] <= v2[0] && v1[1] < v2[1])
			|| (v1[0] <= v2[0] && v1[1] <= v2[1] && v1[2] < v2[2]));
}

CUDA_CALLABLE_MEMBER friend Vec Vec::operator*(const Vec & v1, const Vec & v2) {
	return Vec(v1.data[0] * v2.data[0], v1.data[1] * v2.data[1], v1.data[2] * v2.data[2]);
} // Multiplies two Vecs (elementwise)

CUDA_CALLABLE_MEMBER friend Vec Vec::operator/(const Vec & v, const double x) {
	return Vec(v.data[0] / x, v.data[1] / x, v.data[2] / x);
} //  vector over double

CUDA_CALLABLE_MEMBER friend Vec Vec::operator/(const Vec & v1, const Vec & v2) {
	return Vec(v1.data[0] / v2.data[0], v1.data[1] / v2.data[1], v1.data[2] / v2.data[2]);
} // divides two Vecs (elementwise)

friend std::ostream & Vec::operator << (std::ostream & strm, const Vec & v) {
	return strm << "(" << v[0] << ", " << v[1] << ", " << v[2] << ")";
} // print

CUDA_CALLABLE_MEMBER void Vec::print() {
	printf("(%3f, %3f, %3f)\n", data[0], data[1], data[2]);
}

CUDA_CALLABLE_MEMBER double Vec::norm() const {
	return sqrt(pow(data[0], 2) + pow(data[1], 2) + pow(data[2], 2));
} // gives vector norm

CUDA_CALLABLE_MEMBER double Vec::sum() const {
	return data[0] + data[1] + data[2];
} // sums all components of the vector

CUDA_CALLABLE_MEMBER Vec Vec::normalized() const {
	double l = norm();
	return l > 0 ? (*this)/l : (*this);
}

CUDA_CALLABLE_MEMBER double Vec::dot(const Vec & a, const Vec & b) {
	return (a * b).sum();
}

CUDA_CALLABLE_MEMBER Vec Vec::cross(const Vec &v1, const Vec &v2) {
	return Vec(v1[1] * v2[2] - v1[2] * v2[1], v2[0] * v1[2] - v1[0] * v2[2], v1[0] * v2[1] - v1[1] * v2[0]);
}
