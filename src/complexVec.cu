//
//  complexVec.cpp
//  CUDA Physics
//
//  Created by Salvy Cavicchio on 10/11/20.
//  Copyright © 2020 Salvy Cavicchio. All rights reserved.
//

#include "complexVec.h"

// COMPLEX VECTOR

ComplexVec::ComplexVec() {
	hipDoubleComplex z = make_hipDoubleComplex(0, 0);
    data[0] = z;
    data[1] = z;
   	data[2] = z;
} // default

ComplexVec::ComplexVec(const ComplexVec & v) {
    data[0] = v.data[0];
    data[1] = v.data[1];
    data[2] = v.data[2];
} // copy constructor

ComplexVec::ComplexVec(hipDoubleComplex x, hipDoubleComplex y, hipDoubleComplex z) {
    data[0] = x;
    data[1] = y;
    data[2] = z;
} // initialization from x, y, and z values

ComplexVec & ComplexVec::operator=(const ComplexVec & v) {
    if (this == &v) {
        return *this;
    }

    data[0] = v.data[0];
    data[1] = v.data[1];
    data[2] = v.data[2];

    return *this;
}

ComplexVec & ComplexVec::operator+=(const ComplexVec & v) {
	data[0] = hipCadd(data[0],v.data[0]);
    data[1] = hipCadd(data[1],v.data[1]);
    data[2] = hipCadd(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator-=(const ComplexVec & v) {
	data[0] = hipCsub(data[0],v.data[0]);
    data[1] = hipCsub(data[1],v.data[1]);
    data[2] = hipCsub(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator*=(const ComplexVec & v) {
	data[0] = hipCmul(data[0],v.data[0]);
    data[1] = hipCmul(data[1],v.data[1]);
    data[2] = hipCmul(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator/=(const ComplexVec & v) {
	data[0] = hipCdiv(data[0],v.data[0]);
    data[1] = hipCdiv(data[1],v.data[1]);
    data[2] = hipCdiv(data[2],v.data[2]);
    return *this;
}

ComplexVec & ComplexVec::operator+=(double x) {
	data[0] = hipCadd(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCadd(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCadd(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

ComplexVec & ComplexVec::operator-=(double x) {
	data[0] = hipCsub(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCsub(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCsub(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

ComplexVec & ComplexVec::operator*=(double x) {
	data[0] = hipCmul(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCmul(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCmul(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

ComplexVec & ComplexVec::operator/=(double x) {
	data[0] = hipCdiv(data[0],make_hipDoubleComplex(x, 0));
	data[1] = hipCdiv(data[1],make_hipDoubleComplex(x, 0));
	data[2] = hipCdiv(data[2],make_hipDoubleComplex(x, 0));
	return *this;
}

//CUDA_CALLABLE_MEMBER ComplexVec operator-() const; // returns the negative -z = -a -bi
hipDoubleComplex & ComplexVec::operator [] (int n) {
	if (n < 0 || n >= 3) {
        printf("%s\n", "Out of bounds!");
        return data[0];
    } else {
        return data[n];
    }
}

const hipDoubleComplex & ComplexVec::operator [] (int n) const {
	if (n < 0 || n >= 3) {
        printf("%s\n", "Out of bounds!");
        return data[0];
    } else {
        return data[n];
    }
}
/* WORK IN PROGRESS
bool operator==(const ComplexVec & v1, const ComplexVec & v2) {
	bool real = (hipCreal(v1[0]) == hipCreal(v2[0]) && hipCreal(v1[1]) == hipCreal(v2[1]) && hipCreal(v1[2]) == hipCreal(v2[2]));
	bool imaj = (cuCimaj(v1[0]) == cuCimaj(v2[0]) && cuCimaj(v1[1]) == cuCimaj(v2[1]) && cuCimaj(v1[2]) == cuCimaj(v2[2]));
	return (real && imaj);
}


ComplexVec operator+(const ComplexVec & v1, const ComplexVec & v2) {
	return v1+=v2;
}

ComplexVec operator-(const ComplexVec & v1, const ComplexVec & v2) {
	return v1-=v2;
}

ComplexVec operator*(const ComplexVec & v1, const ComplexVec & v2) {
	return v1*=v2;
}

ComplexVec operator/(const ComplexVec & v1, const ComplexVec & v2) {
	return v1/=v2;
}

std::ostream & operator << (std::ostream & strm, const ComplexVec & v) {
	return strm << "(" << hipCreal(v[0]) << " + " << hipCimag(v[0]) << "i" << ", " << hipCreal(v[1]) << " + " << cuCimaj(v[0]) << "i" << ", " << hipCreal(v[2]) << " + " << hipCimag(v[0]) << "i)";
}


void ComplexVec::print() {
	printf("(%3f + %3fi, %3f + %3fi, %3f + %3fi)\n", hipCreal(data[0]),cuCimaj(data[0]), hipCreal(data[1]),cuCimaj(data[1]), hipCreal(data[2]),cuCimaj(data[2]));
}

hipDoubleComplex ComplexVec::sum() const {
	return make_hipDoubleComplex(hipCadd(hipCadd(data[0],data[1],data[2])));
}
*/
